#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

// Hyperparameters
constexpr float huber_loss_threashold = 10;
constexpr float z_score_trimming_threashold = 2;
constexpr float epsilon = 0.49;
constexpr float learning_rate = 0.01;
constexpr int batch_size = 128;
constexpr int max_iter = 100000;

constexpr int sample_size = 1000;
constexpr int dimension = 6;

template<int block_size> __global__ void kernel(float* const X, float* const y, float* const _w, const clock_t seed) {
    __shared__ float w[dimension];
    __shared__ int indices[batch_size];
    int indices_copy[batch_size];
    __shared__ float residuals[batch_size];
    __shared__ float residuals_copy[batch_size];
    __shared__ float gradient[dimension];
    __shared__ float prev_loss;
    __shared__ float loss;
    __shared__ int index_low;
    __shared__ int index_high;
    //__shared__ bool z_score_trimming_flag_converged;
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);

    // Initialization
    if (threadIdx.x == 0) {
        for (int i = 0; i < dimension; i++) {
            w[i] = 1;
        }
        prev_loss = 0;
    }

    for (int _ = -1; _ < max_iter; _++) {
        // Sample a consecutive batch with random starting index
        indices[threadIdx.x] = ((int)(hiprand_uniform(&state) * sample_size) + threadIdx.x) % sample_size;

        // Calculate residuals
        residuals[threadIdx.x] = -y[indices[threadIdx.x]];
        for (int j = 0; j < dimension; j++) {
            residuals[threadIdx.x] += X[j * sample_size + indices[threadIdx.x]] * w[j];
        }

        // Merge sort residuals and permute the indices accordingly
        for (int i = 1; i < batch_size; i += 2) {
            if (residuals[i] < residuals[i - 1]) {
                const float tmp_float = residuals[i];
                residuals[i] = residuals[i - 1];
                residuals[i - 1] = tmp_float;
                const int tmp_int = indices[i];
                indices[i] = indices[i - 1];
                indices[i - 1] = tmp_int;
            }
        }
        for (int stride = 2; stride < batch_size; stride *= 2) {
            __syncthreads();
            if (threadIdx.x % (stride * 2) == 0) {
                int j = threadIdx.x;
                const int j_end = threadIdx.x + stride;
                if (j_end >= batch_size) {
                    break;
                }
                int k = j_end;
                const int k_end = ((threadIdx.x + stride * 2) > batch_size) ? batch_size : (threadIdx.x + stride * 2);
                int l = threadIdx.x;
                while (j != j_end && k != k_end) {
                    if (residuals[j] < residuals[k]) {
                        residuals_copy[l] = residuals[j];
                        indices_copy[l] = indices[j];
                        j++;
                    }
                    else {
                        residuals_copy[l] = residuals[k];
                        indices_copy[l] = indices[k];
                        k++;
                    }
                    l++;
                }
                if (j == j_end) {
                    for (j = threadIdx.x; j < l; j++) {
                        residuals[j] = residuals_copy[j];
                        indices[j] = indices_copy[j];
                    }
                }
                else {
                    for (k = j_end - 1;k >= j; k--) {
                        residuals[k + k_end - j_end] = residuals[k];
                        indices[k + k_end - j_end] = indices[k];
                    }
                    for (k = threadIdx.x; k < l; k++) {
                        residuals[k] = residuals_copy[k];
                        indices[k] = indices_copy[k];
                    }
                }
            }
        }

        __syncthreads();
        if (threadIdx.x == 0) {
            // Epsilon-trimming
            index_low = 0;
            float abs_residual_low = std::abs(residuals[0]);
            index_high = batch_size - 1;
            float abs_residual_high = std::abs(residuals[batch_size - 1]);
            for (int i = 0; i < (int)(batch_size * epsilon); i++) {
                if (abs_residual_low < abs_residual_high) {
                    residuals[index_high] = 0;
                    index_high--;
                    abs_residual_high = std::abs(residuals[index_high]);
                }
                else {
                    residuals[index_low] = 0;
                    index_low++;
                    abs_residual_low = std::abs(residuals[index_low]);
                }
            }

            // Z-score-trimming
            while (true) {
                float mean = 0;
                for (int i = index_low; i <= index_high; i++) {
                    mean += residuals[i];
                }
                mean /= (index_high - index_low);
                float stdev = 0;
                for (int i = index_low; i <= index_high; i++) {
                    const float diff = residuals[i] - mean;
                    stdev += diff * diff;
                }
                stdev = sqrt(stdev / (index_high - index_low));
                bool flag_converged = true;
                const float threashold_low = mean - stdev * z_score_trimming_threashold;
                while (residuals[index_low] < threashold_low) {
                    residuals[index_low] = 0;
                    index_low++;
                    flag_converged = false;
                }
                const float threashold_high = mean + stdev * z_score_trimming_threashold;
                while (residuals[index_high] > threashold_high) {
                    residuals[index_high] = 0;
                    index_high--;
                    flag_converged = false;
                }
                if (flag_converged) {
                    break;
                }
            }
            loss = 0;
            for (int i = 0; i < dimension; i++) {
                gradient[i] = 0;
            }
        }

        // Calculate Huber Loss and gradient
        __syncthreads();
        const float residual = residuals[threadIdx.x];
        const float abs_residual = std::abs(residual);
        if (abs_residual <= huber_loss_threashold) {
            atomicAdd(&loss, residual * residual / 2);
            for (int j = 0; j < dimension; j++) {
                atomicAdd(gradient + j, residual * X[j * sample_size + indices[threadIdx.x]]);
            }
        }
        else {
            atomicAdd(&loss, abs_residual * huber_loss_threashold - huber_loss_threashold * huber_loss_threashold / 2);
            for (int j = 0; j < dimension; j++) {
                atomicAdd(gradient + j, ((residual > 0) - (residual < 0)) * X[j * sample_size + indices[threadIdx.x]] * huber_loss_threashold);
            }
        }

        // Update weights
        __syncthreads();
        if (threadIdx.x == 0) {
            for (int i = 0; i < dimension; i++) {
                w[i] -= learning_rate * gradient[i] / (index_high - index_low + 1);
            }
        }

        // Check convergence
        __syncthreads();
        if (std::abs((loss - prev_loss) / prev_loss) < 1e-5) {
            break;
        }
        prev_loss = loss;
    }

    // Write to global memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < dimension; i++) {
            _w[i] = w[i];
        }
    }
}

int main(void) {
    float X[sample_size * dimension];
    float y[sample_size];
    float w[dimension];
    srand(clock());

    // Read training data
    FILE* f = fopen("in.txt", "r");
    for (int i = 0; i < sample_size; i++) {
        X[i] = 1;
        for (int j = 1; j < dimension; j++) {
            fscanf(f, "%f", X + j * sample_size + i);
        }
        fscanf(f, "%f", y + i);
    }
    fclose(f);

    // Allocate device memory
    float* device_X, * device_y, * device_w;
    hipMalloc(&device_X, sample_size * dimension * sizeof(float));
    hipMalloc(&device_y, sample_size * sizeof(float));
    hipMalloc(&device_w, dimension * sizeof(float));

    // Copy input to device memory
    hipMemcpy(device_X, X, sample_size * dimension * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, sample_size * sizeof(float), hipMemcpyHostToDevice);

    // Start timing
    clock_t clk = clock();

    kernel<batch_size> << <1, batch_size >> > (device_X, device_y, device_w, clk);

    // Stop timing
    hipDeviceSynchronize();
    clk = clock() - clk;
    printf("CUDA running time:\t%.3fms\n", (double)clk / CLOCKS_PER_SEC * 1000);

    // Copy output to host memory
    hipMemcpy(w, device_w, dimension * sizeof(float), hipMemcpyDeviceToHost);

    // Write the trained weights
    f = fopen("out.txt", "w");
    fprintf(f, "%f", *w);
    for (int i = 1; i < dimension; i++) {
        fprintf(f, " %f", w[i]);
    }
    fclose(f);

    return 0;
}
